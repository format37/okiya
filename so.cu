
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 16

__global__ void test(int *out, int n)
{
    for(int i = 0; i < n; i++) out[i]=round(pow(2,i));
}

int main()
{
    int *a;
    int *out;
    a   = (int*)malloc(sizeof(int) * N);
    hipMalloc((void**)&out, sizeof(int) * N);
    test<<<1, 1>>>(out, N);
    hipMemcpy(a, out, sizeof(int) * N, hipMemcpyDeviceToHost);
    for(int i = 0; i < N; i++) printf("%d\n",a[i]);
}